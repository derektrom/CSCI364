/*
 *Derek Trom
 *HW5 CSCI364
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <iomanip>
#include <cstdio>


__device__ float add(float num){

    float outnum = num + 1;
    return outnum;   
    
}

__global__
void func1(float *xd, float *yd, int n) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = threadId; i < n; i+= stride) {
        yd[i] = add(xd[i]);
    }
}
__global__ 
void createArrays(float *in, float *out, int n){
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = threadId; i < n; i+=stride) {
        in[i] = 1.0f;
        out[i] = 0.0f;
    }
    
}
int main(int argc, char **argv){
    using namespace std;
    if( argc< 3){
        cerr<<"Usage: "<<argv[0]<<" <length of arrays> <num threads/block>"<<endl;
        return 1;
    }
    int threads_per_block = atoi(argv[2]);    
    int sizeOfArray = atoi(argv[1]);
    if (sizeOfArray < 1 or threads_per_block < 1){
        cerr<<"Array length and block size must be > 0"<<endl;
        return 1;
    }
    float *xd, *yd;
    hipMallocManaged(&xd, sizeOfArray*sizeof(float));
    hipMallocManaged(&yd, sizeOfArray*sizeof(float));
    //---------PHASE ONE----------//
    int numBlocks = (sizeOfArray + threads_per_block- 1) / threads_per_block;
    createArrays<<<numBlocks, threads_per_block>>>(xd, yd, sizeOfArray);
    
    func1<<<numBlocks, threads_per_block>>>(xd,yd,sizeOfArray);
    hipDeviceSynchronize();
    float maxError = 0.0f;
    for (int i = 0; i < sizeOfArray; i++)
    {
        maxError = fmax(maxError, fabs(yd[i]-2.0f));
    }
    cout<<"Phase 1"<<endl;
    cout<<endl<<"Array size: "<<sizeOfArray<<endl;
    cout<<"Threads per block: "<<threads_per_block<<endl;
    cout<<"Number of blocks: "<<numBlocks<<endl;
    cout << "Max error: " << maxError << endl;   
    
    //--------Phase 2-------//
    //Use half the number of blocks to get the next number but use
    //the same kernel function
    threads_per_block = threads_per_block/2;
    createArrays<<<numBlocks, threads_per_block>>>(xd, yd, sizeOfArray);
    func1<<<numBlocks, threads_per_block>>>(xd,yd,sizeOfArray);
    hipDeviceSynchronize();
    for (int i = 0; i < sizeOfArray; i++)
    {
        maxError = fmax(maxError, fabs(yd[i]-2.0f));
    }
    cout<<"Phase 2"<<endl;
    cout<<endl<<"Array size: "<<sizeOfArray<<endl;
    cout<<"Threads per block: "<<threads_per_block<<endl;
    cout<<"Number of blocks: "<<numBlocks<<endl;
    cout << "Max error: " << maxError << endl;   
    
    
    hipFree(xd);
    hipFree(yd);
    return 0;
}
